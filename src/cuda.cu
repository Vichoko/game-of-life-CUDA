#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "globals.h"

#define THREADS_PER_BLOCK 8

int* livesArrayActual;
int N;
int size;

int* d_livesArrayActual;
int* d_livesArrayNext;

using namespace std;

__host__ __device__ bool isCellAlive(int* array, int column, int row){
	return array[row*COLUMNS + column];

}
__device__ void setCellAlive(int* array, int column, int row){
	array[row*COLUMNS + column] = 1;
}
__device__ void setCellDead(int* array, int column, int row){
	array[row*COLUMNS + column] = 0;
}
__device__ int countAliveNeighbors(int* livesArray, int column, int row){
	int neighborColumn;
	int neighborRow;
	int aliveNeighbors = 0;
	for (int x = -1; x <= 1; x++){
		for (int y = -1; y <= 1; y++){
			if (x == 0 && y == 0) // itself bypass
				continue;
			neighborColumn = column+x;
			neighborRow = row+y;

			// edge conditions
			if (neighborColumn < 0){
				neighborColumn = COLUMNS-1;
			} else if (neighborColumn > COLUMNS-1){
				neighborColumn = 0;
			} if (neighborRow < 0){
				neighborRow = ROWS-1;
			} else if (neighborRow > ROWS-1){
				neighborRow = 0;
			}

			if (isCellAlive(livesArray, neighborColumn, neighborRow)){
				aliveNeighbors++;
			}
		}
	}
	return aliveNeighbors;
}


int* generateInitialLives(int seed, int aliveCellsSize){
	int* cellIndexes = (int*) malloc(sizeof(int)*aliveCellsSize);

	srand(seed);
	for (int i = 0; i < aliveCellsSize; i++){
		int cellIndex = rand() % (COLUMNS*ROWS);
		bool validRandom = true;

		for (int j = 0; j < i; j++){
			if (cellIndexes[j] == cellIndex){
				// need to pick another random number
				i--;
				validRandom = false;
				break;
			}
		}
		if (validRandom)
			cellIndexes[i] = cellIndex;
	}
	return cellIndexes;

}


void swapLivesArrays(int** livesArrayActual, int** livesArrayNext){
	int* aux;
	aux = *livesArrayActual;
	*livesArrayActual = *livesArrayNext;
	*livesArrayNext = aux;
}


int* init_game_data(){
	N = COLUMNS*ROWS;
	size = sizeof(int)*N;

	livesArrayActual = (int*) malloc(size);

	hipError_t code = hipSuccess;
	hipMalloc((void **)&d_livesArrayActual, size);
	code = hipGetLastError();
	if (code != hipSuccess){
		printf("error alocating d_livesArrayActual\n");
	}
	hipMalloc((void **)&d_livesArrayNext, size);
	code = hipGetLastError();
	if (code != hipSuccess){
		printf("error alocating d_livesArrayNext\n");
	}

	int initialAliveCellsSize = (int) COLUMNS*ROWS*0.3;
	int* initialAliveCells = generateInitialLives(1, initialAliveCellsSize);
	for (int i = 0; i < initialAliveCellsSize; i++){
		// initial set up of lives
		livesArrayActual[initialAliveCells[i]] = 1;
	}
	free(initialAliveCells);
	hipMemcpy(d_livesArrayActual, livesArrayActual, size, hipMemcpyHostToDevice);
	code = hipGetLastError();
	if (code != hipSuccess){
		printf("error copying d_livesArrayActual\n");
	}
	return livesArrayActual;
}

__global__ void refreshLife(int* livesArrayActual, int* livesArrayNext) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int row = floorf(index / COLUMNS);
	int column = index - row*COLUMNS;

	int aliveNeighbors = countAliveNeighbors(livesArrayActual, column, row);
	if ((isCellAlive(livesArrayActual, column, row) && (aliveNeighbors == 2 || aliveNeighbors == 3)) ||
			(!isCellAlive(livesArrayActual, column, row) && (aliveNeighbors == 3 || aliveNeighbors == 6))){
		// life condition
		setCellAlive(livesArrayNext, column, row);
	} else {
		// death condition
		setCellDead(livesArrayNext, column, row);
	}
}

/** 
* Ejecuta kernel y retorna tiempo (en ms) total de procesamiento del kernel.
**/
float kernel_wrapper(){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipError_t code = hipSuccess;

	hipEventRecord(start);
	refreshLife<<<(N + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_livesArrayActual, d_livesArrayNext);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	// error check
	code = hipGetLastError();
	if (code != hipSuccess){
		printf("error kernel refreshLife %s\n",  hipGetErrorString(code));
		exit(-1);
	}
	return milliseconds;
}

int* fetch_gpu_data(){
	hipError_t code = hipSuccess;
	hipMemcpy(livesArrayActual, d_livesArrayNext, size, hipMemcpyDeviceToHost);
	code = hipGetLastError();
	if (code != hipSuccess){
		printf("error copying livesArrayActual %s\n",  hipGetErrorString(code));
	}
	hipDeviceSynchronize();
	swapLivesArrays(&d_livesArrayActual, &d_livesArrayNext);
	return livesArrayActual;
}
void free_cuda_resources(){
	hipFree(d_livesArrayActual); hipFree(d_livesArrayNext);

}






